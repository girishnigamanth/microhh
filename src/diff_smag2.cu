#include "hip/hip_runtime.h"
/*
 * MicroHH
 * Copyright (c) 2011-2020 Chiel van Heerwaarden
 * Copyright (c) 2011-2020 Thijs Heus
 * Copyright (c) 2014-2020 Bart van Stratum
 *
 * This file is part of MicroHH
 *
 * MicroHH is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.

 * MicroHH is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.

 * You should have received a copy of the GNU General Public License
 * along with MicroHH.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <cstdio>
#include <cmath>
#include <algorithm>
#include <iostream>

#include "grid.h"
#include "fields.h"
#include "master.h"
#include "diff_smag2.h"
#include "boundary.h"
#include "boundary_surface.h"
#include "defines.h"
#include "constants.h"
#include "thermo.h"
#include "tools.h"
#include "stats.h"
#include "monin_obukhov.h"
#include "fast_math.h"

#include "diff_kernels.cuh"

namespace
{
    namespace most = Monin_obukhov;
    namespace fm = Fast_math;
    namespace dk = Diff_kernels_g;

    template<typename TF, Surface_model surface_model> __global__
    void evisc_g(
            TF* __restrict__ evisc,
            TF* __restrict__ N2,
            TF* __restrict__ bgradbot,
            TF* __restrict__ mlen0,
            TF* __restrict__ z0m,
            TF* __restrict__ z,
            const TF tPri,
            const int istart, const int iend,
            const int jstart, const int jend,
            const int kstart, const int kend,
            const int jj,     const int kk)
    {
        const int i = blockIdx.x*blockDim.x + threadIdx.x + istart;
        const int j = blockIdx.y*blockDim.y + threadIdx.y + jstart;
        const int k = blockIdx.z + kstart;

        const TF n_mason = TF(2);

        if (i < iend && j < jend && k < kend)
        {
            const int ij  = i + j*jj;
            const int ijk = i + j*jj + k*kk;

            if (k == kstart && surface_model == Surface_model::Enabled)
            {
                // calculate smagorinsky constant times filter width squared, use wall damping according to Mason
                TF RitPrratio = bgradbot[ij] / evisc[ijk] * tPri;
                RitPrratio = fmin(RitPrratio, TF(1.-Constants::dsmall));

                const TF mlen = std::pow(TF(1.)/(TF(1.)/mlen0[k] + TF(1.)/(std::pow(Constants::kappa<TF>*(z[kstart]+z0m[ij]), n_mason))), TF(1.)/n_mason);
                evisc[ijk] = fm::pow2(mlen) * sqrt(evisc[ijk] * (TF(1.)-RitPrratio));
            }
            else if (surface_model == Surface_model::Enabled)
            {
                // Add the buoyancy production to the TKE
                TF RitPrratio = N2[ijk] / evisc[ijk] * tPri;
                RitPrratio = fmin(RitPrratio, TF(1.-Constants::dsmall));

                // Mason mixing length
                const TF mlen = std::pow(TF(1.)/(TF(1.)/mlen0[k] + TF(1.)/(std::pow(Constants::kappa<TF>*(z[k]+z0m[ij]), n_mason))), TF(1.)/n_mason);
                evisc[ijk] = fm::pow2(mlen) * sqrt(evisc[ijk] * (TF(1.)-RitPrratio));
            }
            else
            {
                // calculate smagorinsky constant times filter width squared, use wall damping according to Mason
                TF RitPrratio = N2[ijk] / evisc[ijk] * tPri;
                RitPrratio = fmin(RitPrratio, TF(1.-Constants::dsmall));
                evisc[ijk] = fm::pow2(mlen0[k]) * sqrt(evisc[ijk] * (TF(1.)-RitPrratio));
            }
        }
    }

    template<typename TF> __global__
    void evisc_neutral_g(
            TF* __restrict__ evisc,
            TF* __restrict__ z0m,
            TF* __restrict__ z,
            TF* __restrict__ mlen0,
            const int istart, const int jstart, const int kstart,
            const int iend, const int jend, const int kend,
            const int jj, const int kk)
    {
        const int i = blockIdx.x*blockDim.x + threadIdx.x + istart;
        const int j = blockIdx.y*blockDim.y + threadIdx.y + jstart;
        const int k = blockIdx.z + kstart;

        const TF n_mason = TF(2);

        if (i < iend && j < jend && k < kend)
        {
            const int ijk = i + j*jj + k*kk;
            const int ij = i + j*jj;

            const TF mlen = std::pow(TF(1.)/(TF(1.)/mlen0[k] + TF(1.)/(std::pow(Constants::kappa<TF>*(z[k]+z0m[ij]), n_mason))), TF(1.)/n_mason);
            evisc[ijk] = fm::pow2(mlen) * sqrt(evisc[ijk]);
        }
    }

    template<typename TF> __global__
    void evisc_neutral_vandriest_g(
            TF* __restrict__ evisc,
            const TF* __restrict__ u, const TF* __restrict__ v,
            const TF* __restrict__ mlen_smag,
            const TF* __restrict__ z, const TF* __restrict__ dzhi,
            const TF zsize, const TF visc,
            const int istart, const int jstart, const int kstart,
            const int iend, const int jend, const int kend,
            const int jj, const int kk)

    {
        const int i = blockIdx.x*blockDim.x + threadIdx.x + istart;
        const int j = blockIdx.y*blockDim.y + threadIdx.y + jstart;
        const int k = blockIdx.z + kstart;

        const TF A_vandriest = TF(26.);

        if (i < iend && j < jend && k < kend)
        {
            const int ijk = i + j*jj + k*kk;
            const int ijk_bot = i + j*jj + kstart*kk;
            const int ijk_top = i + j*jj + kend*kk;

            const TF u_tau_bot = pow(
                    fm::pow2( visc*(u[ijk_bot] - u[ijk_bot-kk] )*dzhi[kstart] )
                  + fm::pow2( visc*(v[ijk_bot] - v[ijk_bot-kk] )*dzhi[kstart] ), TF(0.25) );
            const TF u_tau_top = pow(
                    fm::pow2( visc*(u[ijk_top] - u[ijk_top-kk] )*dzhi[kend] )
                  + fm::pow2( visc*(v[ijk_top] - v[ijk_top-kk] )*dzhi[kend] ), TF(0.25) );

            const TF fac_bot = TF(1.) - exp( -(       z[k] *u_tau_bot) / (A_vandriest*visc) );
            const TF fac_top = TF(1.) - exp( -((zsize-z[k])*u_tau_top) / (A_vandriest*visc) );

            const TF fac = min(fac_bot, fac_top);

            evisc[ijk] = fm::pow2(fac * mlen_smag[k]) * sqrt(evisc[ijk]);
        }
    }

    template<typename TF> __global__
    void calc_ghostcells_evisc(
            TF* __restrict__ evisc,
            const int icells, const int jcells,
            const int kstart, const int kend,
            const int jj, const int kk)
    {
        const int i = blockIdx.x*blockDim.x + threadIdx.x;
        const int j = blockIdx.y*blockDim.y + threadIdx.y;

        if (i < icells && j < jcells)
        {
            const int kb = kstart;
            const int kt = kend-1;

            const int ijkb = i + j*jj + kb*kk;
            const int ijkt = i + j*jj + kt*kk;

            evisc[ijkb-kk] = evisc[ijkb];
            evisc[ijkt+kk] = evisc[ijkt];
        }
    }
}

/* Calculate the mixing length (mlen) offline, and put on GPU */
#ifdef USECUDA
template<typename TF>
void Diff_smag2<TF>::prepare_device(Boundary<TF>& boundary)
{
    auto& gd = grid.get_grid_data();

    std::vector<TF> mlen(gd.kcells);

    if (boundary.get_switch() == "default")
    {
        for (int k=0; k<gd.kcells; ++k)
            mlen[k] = cs * pow(gd.dx*gd.dy*gd.dz[k], 1./3.);
    }
    else
    {
        const TF n_mason = TF(2);
        for (int k=0; k<gd.kcells; ++k)
            mlen[k] = std::pow(cs * std::pow(gd.dx*gd.dy*gd.dz[k], TF(1./3.)), n_mason);
    }

    const int nmemsize = gd.kcells*sizeof(TF);
    cuda_safe_call(hipMalloc(&mlen_g, nmemsize));
    cuda_safe_call(hipMemcpy(mlen_g, mlen.data(), nmemsize, hipMemcpyHostToDevice));
}

template<typename TF>
void Diff_smag2<TF>::clear_device()
{
    cuda_safe_call(hipFree(mlen_g));
}
#endif

#ifdef USECUDA
template<typename TF>
void Diff_smag2<TF>::exec_viscosity(Stats<TF>&, Thermo<TF>& thermo)
{
    auto& gd = grid.get_grid_data();

    const int blocki = gd.ithread_block;
    const int blockj = gd.jthread_block;
    const int gridi  = gd.imax/blocki + (gd.imax%blocki > 0);
    const int gridj  = gd.jmax/blockj + (gd.jmax%blockj > 0);

    dim3 gridGPU (gridi, gridj, gd.kcells);
    dim3 blockGPU(blocki, blockj, 1);

    // Contain the full icells and jcells in this grid.
    const int grid2di  = gd.icells/blocki + (gd.icells%blocki > 0);
    const int grid2dj  = gd.jcells/blockj + (gd.jcells%blockj > 0);

    dim3 grid2dGPU (grid2di, grid2dj);
    dim3 block2dGPU(blocki, blockj);

    // Use surface model.
    if (boundary.get_switch() != "default")
    {
        TF* z0m_g   = boundary.get_z0m_g();

        // Get MO gradients velocity:
        TF* dudz_g  = boundary.get_dudz_g();
        TF* dvdz_g  = boundary.get_dvdz_g();

        // Calculate total strain rate
        dk::calc_strain2_g<TF, Surface_model::Enabled><<<gridGPU, blockGPU>>>(
            fields.sd.at("evisc")->fld_g,
            fields.mp.at("u")->fld_g,
            fields.mp.at("v")->fld_g,
            fields.mp.at("w")->fld_g,
            dudz_g, dvdz_g,
            gd.dzi_g, gd.dzhi_g,
            gd.dxi, gd.dyi,
            gd.istart, gd.iend,
            gd.jstart, gd.jend,
            gd.kstart, gd.kend,
            gd.icells, gd.ijcells);
        cuda_check_error();

        if (thermo.get_switch() == Thermo_type::Disabled)
        {
            // Start with retrieving the stability information
            evisc_neutral_g<TF><<<gridGPU, blockGPU>>>(
                fields.sd.at("evisc")->fld_g,
                z0m_g, gd.z_g, mlen_g,
                gd.istart, gd.jstart, gd.kstart,
                gd.iend,   gd.jend,   gd.kend,
                gd.icells, gd.ijcells);
            cuda_check_error();
        }
        else
        {
            // Assume buoyancy calculation is needed
            auto tmp1 = fields.get_tmp_g();
            thermo.get_thermo_field_g(*tmp1, "N2", false);

            // Get MO gradient buoyancy:
            TF* dbdz_g  = boundary.get_dbdz_g();

            // Calculate eddy viscosity
            TF tPri = 1./tPr;

            evisc_g<TF, Surface_model::Enabled><<<gridGPU, blockGPU>>>(
                fields.sd.at("evisc")->fld_g,
                tmp1->fld_g, dbdz_g,
                mlen_g, z0m_g, gd.z_g,
                tPri,
                gd.istart, gd.iend,
                gd.jstart, gd.jend,
                gd.kstart, gd.kend,
                gd.icells, gd.ijcells);
            cuda_check_error();

            fields.release_tmp_g(tmp1);
        }

        boundary_cyclic.exec_g(fields.sd.at("evisc")->fld_g);
    }
    // Do not use surface model.
    else
    {
        // Calculate total strain rate
        dk::calc_strain2_g<TF, Surface_model::Disabled><<<gridGPU, blockGPU>>>(
            fields.sd.at("evisc")->fld_g,
            fields.mp.at("u")->fld_g,
            fields.mp.at("v")->fld_g,
            fields.mp.at("w")->fld_g,
            nullptr, nullptr,
            gd.dzi_g, gd.dzhi_g,
            gd.dxi, gd.dyi,
            gd.istart, gd.iend,
            gd.jstart, gd.jend,
            gd.kstart, gd.kend,
            gd.icells, gd.ijcells);
        cuda_check_error();

        // start with retrieving the stability information
        if (thermo.get_switch() == Thermo_type::Disabled)
        {
            evisc_neutral_vandriest_g<TF><<<gridGPU, blockGPU>>>(
                fields.sd.at("evisc")->fld_g,
                fields.mp.at("u")->fld_g,
                fields.mp.at("v")->fld_g,
                mlen_g, gd.z_g, gd.dzhi_g,
                gd.zsize, fields.visc,
                gd.istart, gd.jstart, gd.kstart,
                gd.iend, gd.jend, gd.kend,
                gd.icells, gd.ijcells);
            cuda_check_error();
        }
        // assume buoyancy calculation is needed
        else
        {
            // store the buoyancyflux in datafluxbot of tmp1
            auto tmp1 = fields.get_tmp_g();
            thermo.get_buoyancy_fluxbot_g(*tmp1);
            // As we only use the fluxbot field of tmp1 we store the N2 in the interior.
            thermo.get_thermo_field_g(*tmp1, "N2", false);

            // Calculate eddy viscosity
            TF tPri = 1./tPr;

            evisc_g<TF, Surface_model::Enabled><<<gridGPU, blockGPU>>>(
                fields.sd.at("evisc")->fld_g,
                tmp1->fld_g, nullptr,
                mlen_g, nullptr, gd.z_g,
                tPri,
                gd.istart, gd.iend,
                gd.jstart, gd.jend,
                gd.kstart, gd.kend,
                gd.icells, gd.ijcells);

            cuda_check_error();

            fields.release_tmp_g(tmp1);
        }

        boundary_cyclic.exec_g(fields.sd.at("evisc")->fld_g);
        calc_ghostcells_evisc<TF><<<grid2dGPU, block2dGPU>>>(
                fields.sd.at("evisc")->fld_g,
                gd.icells, gd.jcells,
                gd.kstart, gd.kend,
                gd.icells, gd.ijcells);
    }
}
#endif

#ifdef USECUDA
template<typename TF>
void Diff_smag2<TF>::exec(Stats<TF>& stats)
{
    auto& gd = grid.get_grid_data();

    const int blocki = gd.ithread_block;
    const int blockj = gd.jthread_block;
    const int gridi  = gd.imax/blocki + (gd.imax%blocki > 0);
    const int gridj  = gd.jmax/blockj + (gd.jmax%blockj > 0);

    dim3 gridGPU (gridi, gridj, gd.kmax);
    dim3 blockGPU(blocki, blockj, 1);

    const TF dxidxi = TF(1)/(gd.dx * gd.dx);
    const TF dyidyi = TF(1)/(gd.dy * gd.dy);
    const TF tPri = TF(1)/tPr;

    // Do not use surface model.
    if (boundary.get_switch() == "default")
    {
        dk::diff_uvw_g<TF, Surface_model::Disabled><<<gridGPU, blockGPU>>>(
                fields.mt.at("u")->fld_g,
                fields.mt.at("v")->fld_g,
                fields.mt.at("w")->fld_g,
                fields.sd.at("evisc")->fld_g,
                fields.mp.at("u")->fld_g,
                fields.mp.at("v")->fld_g,
                fields.mp.at("w")->fld_g,
                fields.mp.at("u")->flux_bot_g,
                fields.mp.at("u")->flux_top_g,
                fields.mp.at("v")->flux_bot_g,
                fields.mp.at("v")->flux_top_g,
                gd.dzi_g,
                gd.dzhi_g,
                fields.rhoref_g,
                fields.rhorefh_g,
                gd.dxi,
                gd.dyi,
                fields.visc,
                gd.istart, gd.iend,
                gd.jstart, gd.jend,
                gd.kstart, gd.kend,
                gd.icells, gd.ijcells);

        cuda_check_error();

        for (auto it : fields.st)
        {
            dk::diff_c_g<TF, Surface_model::Disabled><<<gridGPU, blockGPU>>>(
                    it.second->fld_g,
                    fields.sp.at(it.first)->fld_g,
                    fields.sd.at("evisc")->fld_g,
                    fields.sp.at(it.first)->flux_bot_g,
                    fields.sp.at(it.first)->flux_top_g,
                    gd.dzi_g,
                    gd.dzhi_g,
                    fields.rhoref_g,
                    fields.rhorefh_g,
                    dxidxi,
                    dyidyi,
                    tPri,
                    fields.sp.at(it.first)->visc,
                    gd.istart, gd.iend,
                    gd.jstart, gd.jend,
                    gd.kstart, gd.kend,
                    gd.icells, gd.ijcells);
        }
        cuda_check_error();
    }
    // Use surface model.
    else
    {
        dk::diff_uvw_g<TF, Surface_model::Enabled><<<gridGPU, blockGPU>>>(
                fields.mt.at("u")->fld_g,
                fields.mt.at("v")->fld_g,
                fields.mt.at("w")->fld_g,
                fields.sd.at("evisc")->fld_g,
                fields.mp.at("u")->fld_g,
                fields.mp.at("v")->fld_g,
                fields.mp.at("w")->fld_g,
                fields.mp.at("u")->flux_bot_g,
                fields.mp.at("u")->flux_top_g,
                fields.mp.at("v")->flux_bot_g,
                fields.mp.at("v")->flux_top_g,
                gd.dzi_g,
                gd.dzhi_g,
                fields.rhoref_g,
                fields.rhorefh_g,
                gd.dxi,
                gd.dyi,
                fields.visc,
                gd.istart, gd.iend,
                gd.jstart, gd.jend,
                gd.kstart, gd.kend,
                gd.icells, gd.ijcells);
        cuda_check_error();

        for (auto it : fields.st)
            dk::diff_c_g<TF, Surface_model::Enabled><<<gridGPU, blockGPU>>>(
                    it.second->fld_g,
                    fields.sp.at(it.first)->fld_g,
                    fields.sd.at("evisc")->fld_g,
                    fields.sp.at(it.first)->flux_bot_g,
                    fields.sp.at(it.first)->flux_top_g,
                    gd.dzi_g,
                    gd.dzhi_g,
                    fields.rhoref_g,
                    fields.rhorefh_g,
                    dxidxi,
                    dyidyi,
                    tPri,
                    fields.sp.at(it.first)->visc,
                    gd.istart, gd.iend,
                    gd.jstart, gd.jend,
                    gd.kstart, gd.kend,
                    gd.icells, gd.ijcells);
        cuda_check_error();
    }

    hipDeviceSynchronize();
    stats.calc_tend(*fields.mt.at("u"), tend_name);
    stats.calc_tend(*fields.mt.at("v"), tend_name);
    stats.calc_tend(*fields.mt.at("w"), tend_name);
    for (auto it : fields.st)
        stats.calc_tend(*it.second, tend_name);

}
#endif

#ifdef USECUDA
template<typename TF>
unsigned long Diff_smag2<TF>::get_time_limit(unsigned long idt, double dt)
{
    auto& gd = grid.get_grid_data();

    const int blocki = gd.ithread_block;
    const int blockj = gd.jthread_block;
    const int gridi  = gd.imax/blocki + (gd.imax%blocki > 0);
    const int gridj  = gd.jmax/blockj + (gd.jmax%blockj > 0);

    dim3 gridGPU (gridi, gridj, gd.kmax);
    dim3 blockGPU(blocki, blockj, 1);

    const TF dxidxi = TF(1)/(gd.dx * gd.dx);
    const TF dyidyi = TF(1)/(gd.dy * gd.dy);
    const TF tPrfac_i = TF(1)/std::min(TF(1.), tPr);

    auto tmp1 = fields.get_tmp_g();

    // Calculate dnmul in tmp1 field
    dk::calc_dnmul_g<<<gridGPU, blockGPU>>>(
            tmp1->fld_g,
            fields.sd.at("evisc")->fld_g,
            gd.dzi_g,
            tPrfac_i,
            dxidxi, dyidyi,
            gd.istart, gd.iend,
            gd.jstart, gd.jend,
            gd.kstart, gd.kend,
            gd.icells, gd.ijcells);
    cuda_check_error();

    // Get maximum from tmp1 field
    double dnmul = field3d_operators.calc_max_g(tmp1->fld_g);
    dnmul = std::max(Constants::dsmall, dnmul);

    const unsigned long idtlim = idt * dnmax/(dnmul*dt);

    fields.release_tmp_g(tmp1);

    return idtlim;
}
#endif

#ifdef USECUDA
template<typename TF>
double Diff_smag2<TF>::get_dn(double dt)
{
    auto& gd = grid.get_grid_data();

    const int blocki = gd.ithread_block;
    const int blockj = gd.jthread_block;
    const int gridi  = gd.imax/blocki + (gd.imax%blocki > 0);
    const int gridj  = gd.jmax/blockj + (gd.jmax%blockj > 0);

    dim3 gridGPU (gridi, gridj, gd.kmax);
    dim3 blockGPU(blocki, blockj, 1);

    const TF dxidxi = TF(1)/(gd.dx * gd.dx);
    const TF dyidyi = TF(1)/(gd.dy * gd.dy);
    const TF tPrfac_i = TF(1)/std::min(TF(1.), tPr);

    // Calculate dnmul in tmp1 field
    auto dnmul_tmp = fields.get_tmp_g();

    dk::calc_dnmul_g<<<gridGPU, blockGPU>>>(
        dnmul_tmp->fld_g,
        fields.sd.at("evisc")->fld_g,
        gd.dzi_g,
        tPrfac_i,
        dxidxi, dyidyi,
        gd.istart, gd.iend,
        gd.jstart, gd.jend,
        gd.kstart, gd.kend,
        gd.icells, gd.ijcells);
    cuda_check_error();

    // Get maximum from tmp1 field
    // CvH This is odd, because there might be need for calc_max in CPU version.
    double dnmul = field3d_operators.calc_max_g(dnmul_tmp->fld_g);

    fields.release_tmp_g(dnmul_tmp);

    return dnmul*dt;
}
#endif


#ifdef FLOAT_SINGLE
template class Diff_smag2<float>;
#else
template class Diff_smag2<double>;
#endif
